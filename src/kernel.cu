#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cutil_inline.h>
#include <stdio.h>

#include "kernel.h"


#define __DEBUG_MODE_ON__
#define __BENCHMARK_MODE_ON__

#define THREADS_PER_BLOCK 256
#define ROUNDS_NUMBER 24
#define WORDS_NUMBER 25

#define index(x, y) (((x)%5)+5*((y)%5))

// NVCC Bug
//#define ROL64(a, offset) ((offset != 0) ? ((((UINT64)a) << offset) ^ (((UINT64)a) >> (64-offset))) : a)
__device__ inline UINT64 ROL64(UINT64 a, unsigned int offset)
{
	const int _offset = offset;
	return ((offset != 0) ? ((a << _offset) ^ (a >> (64-offset))) : a);
}


UINT64 *buffer_d;
UINT64 *buffer1_d;
UINT64 *buffer2_d;
UINT64 *state_d;

unsigned int threads_number;
size_t size;
size_t size_actual;

hipEvent_t startEvent;
hipEvent_t stopEvent;


__constant__ UINT64 KeccakRoundConstants[ROUNDS_NUMBER];
__constant__ unsigned int KeccakRhoOffsets[WORDS_NUMBER];


/*
 *
 */
__global__ void kernel(UINT64 *messages_d, UINT64 *state_d)
{
	int offset = WORDS_NUMBER * (threadIdx.x + blockIdx.x * blockDim.x);
	unsigned int i, x, y, round_number;
	UINT64 A[WORDS_NUMBER], tempA[WORDS_NUMBER], C[5], D[5];
	
	// Absorbing
	for(i = 0; i < WORDS_NUMBER; i++)
        	A[i] = state_d[offset + i] ^ messages_d[offset + i];

    	for(round_number = 0; round_number < ROUNDS_NUMBER; round_number++) {
		// Theta
		for(x=0; x<5; x++) {
			C[x] = 0; 
			for(y=0; y<5; y++) 
				C[x] ^= A[index(x, y)];
			D[x] = ROL64(C[x], 1);
		}
		for(x=0; x<5; x++)
			for(y=0; y<5; y++)
				A[index(x, y)] ^= D[(x+1)%5] ^ C[(x+4)%5];

        	// Rho
		for(x=0; x<5; x++) 
			for(y=0; y<5; y++)
				A[index(x, y)] = ROL64(A[index(x, y)], KeccakRhoOffsets[index(x, y)]);

		// Pi
        	for(x=0; x<5; x++) for(y=0; y<5; y++)
			tempA[index(x, y)] = A[index(x, y)];
		for(x=0; x<5; x++) for(y=0; y<5; y++)
			A[index(0*x+1*y, 2*x+3*y)] = tempA[index(x, y)];
		
        	// Chi
        	for(y=0; y<5; y++) { 
			for(x=0; x<5; x++)
				C[x] = A[index(x, y)] ^ ((~A[index(x+1, y)]) & A[index(x+2, y)]);
			for(x=0; x<5; x++)
				A[index(x, y)] = C[x];
		}
		
        	// Iota
		A[index(0, 0)] ^= KeccakRoundConstants[round_number];
    }
    
    for(i = 0; i < WORDS_NUMBER; i++)
        state_d[offset + i] = A[i];
}


/*
 *
 */
__global__ void kernel_optimixed(UINT64 *messages_d, UINT64 *state_d)
{
	int offset = WORDS_NUMBER * (threadIdx.x + blockIdx.x * blockDim.x);
	unsigned int x, y, round_number;
	UINT64 A[WORDS_NUMBER], tempA[WORDS_NUMBER], C[5], D[5];
	
	// Absorbing
	for(x = 0; x < WORDS_NUMBER; x++)
        	A[x] = state_d[offset + x] ^ messages_d[offset + x];

    	for(round_number = 0; round_number < ROUNDS_NUMBER; round_number++) {
		// Theta
		for(x=0; x<5; x++) {
			C[x] = 0; 
			for(y=0; y<5; y++) 
				C[x] = C[x] ^ A[index(x, y)];
			D[x] = ROL64(C[x], 1);
		}
		for(x=0; x<5; x++)
			for(y=0; y<5; y++)
				A[index(x, y)] = A[index(x,y)] ^ D[(x+1)%5] ^ C[(x+4)%5];

        	// Rho
		for(x=0; x<5; x++) 
			for(y=0; y<5; y++)
				A[index(x, y)] = ROL64(A[index(x, y)], KeccakRhoOffsets[index(x, y)]);

		// Pi
        	for(x=0; x<5; x++) for(y=0; y<5; y++)
			tempA[index(x, y)] = A[index(x, y)];
		for(x=0; x<5; x++) for(y=0; y<5; y++)
			A[index(0*x+1*y, 2*x+3*y)] = tempA[index(x, y)];
		
        	// Chi
        	for(y=0; y<5; y++) { 
			for(x=0; x<5; x++)
				C[x] = A[index(x, y)] ^ ((~A[index(x+1, y)]) & A[index(x+2, y)]);
			for(x=0; x<5; x++)
				A[index(x, y)] = C[x];
		}
		
        	// Iota
		A[index(0, 0)] = A[index(0,0)] ^ KeccakRoundConstants[round_number];
    }
    
    for(x = 0; x < WORDS_NUMBER; x++)
        state_d[offset + x] = A[x];
}







/*
 *
 */
void launch_kernel(unsigned long long *messages_h, unsigned int token_number)
{
	dim3 threads_per_block(THREADS_PER_BLOCK);
	int num_blocks = threads_number/THREADS_PER_BLOCK;

	if(token_number%2 == 0)
		buffer_d = buffer1_d;
	else
		buffer_d = buffer2_d;

	// Copy messages_h into buffer_d
	cutilSafeCall( hipMemcpy(buffer_d, messages_h, size_actual, hipMemcpyHostToDevice) );

	// Wait old kernel termination
	hipDeviceSynchronize();

	// Launch timer
	if(token_number == 0)
	{
		cutilSafeCall(hipEventRecord(startEvent, 0));
	}

	// Launch new kernel
	kernel_optimixed<<<num_blocks, threads_per_block>>>(buffer_d, state_d);
}


/*
 *
 */
int init_cuda(unsigned int t, UINT64 *krc, unsigned int *kro)
{
	int dev_ID;
	hipDeviceProp_t device_prop;

	// Get best device properties
	dev_ID = cutGetMaxGflopsDeviceId();
	hipGetDeviceProperties(&device_prop, dev_ID);
	#ifdef __DEBUG_MODE_ON__
	printf("*\nMax Gflops Device: \"%s\"\n", device_prop.name);
	printf("\tCUDA Capability:                               %d.%d\n", device_prop.major, device_prop.minor);
	printf("\tTotal amount of Global Memory:                 %llu bytes\n", (UINT64) device_prop.totalGlobalMem);
	//printf("\tMultiprocessor x Cores/MP = Cores:             %d (MP) x %d (Cores/MP) = %d (Cores)\n", device_prop.multiProcessorCount, ConvertSMVer2Cores(deviceProp.major, deviceProp.minor), ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount);
	printf("\tTotal number of registers available per block: %d\n", device_prop.regsPerBlock);
	printf("\tMaximum number of threads per block:           %d\n", device_prop.maxThreadsPerBlock);
	printf("*\n");
	#endif

	// Set device
        hipSetDevice(dev_ID);

	// Set the number of actual threads
	// In order to avoid control instructions inside the kernel, the number of threads is chooses...
	threads_number = ((t%THREADS_PER_BLOCK == 0) ? (t) : (t/THREADS_PER_BLOCK + 1)*THREADS_PER_BLOCK);

	// Meaningfull part of the memory
	size_actual = 25*t*sizeof(UINT64); 

	// Whole memory
	size = 25*threads_number*sizeof(UINT64); 
	
	// Initialize round constants
	cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL("KeccakRoundConstants"), krc, ROUNDS_NUMBER*sizeof(UINT64), 0, hipMemcpyHostToDevice) );
	
	// Initialize rho offsets
	cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL("KeccakRhoOffsets"), kro, WORDS_NUMBER*sizeof(unsigned int), 0, hipMemcpyHostToDevice) );

	// Create timers
	cutilSafeCall( hipEventCreate(&startEvent) );
      	cutilSafeCall( hipEventCreate(&stopEvent) );
	
	return 0;
}



/*
 * Allocate and zero initialize GPU memory
 */
int alloc_memory()
{	
	// Allocate GPU memory buffer 1
	cutilSafeCall( hipMalloc((void**) &buffer1_d, size) );

	// Allocate GPU memory buffer 2
	cutilSafeCall( hipMalloc((void**) &buffer2_d, size) );

	// Allocate GPU memory state
	cutilSafeCall( hipMalloc((void**) &state_d, size) );


	// Zero init
	cutilSafeCall ( hipMemset(buffer1_d, 0, size) );
	cutilSafeCall ( hipMemset(buffer2_d, 0, size) );
	cutilSafeCall ( hipMemset(state_d, 0, size) );

	return 0;
}


/*
 *
 */
int free_memory()
{
	// Deallocate GPU memory buffer 1
	cutilSafeCall(hipFree(buffer1_d));

	// Deallocate GPU memory buffer 2
	cutilSafeCall(hipFree(buffer2_d));

	// Deallocate GPU memory state
	cutilSafeCall(hipFree(state_d));

	return 0;

}


/*
 *
 */
int get_state(UINT64 *state_h)
{
	float milliseconds;

	// Check kernel termination
	cutilSafeCall(hipDeviceSynchronize());

	// Stop timer
	cutilSafeCall(hipEventRecord(stopEvent, 0));
	cutilSafeCall(hipEventSynchronize(stopEvent));
	cutilSafeCall( hipEventElapsedTime(&milliseconds, startEvent, stopEvent));

	#ifdef __BENCHMARK_MODE_ON__
	printf("*\nGPU time: %.3f ms\n*\n", milliseconds);
	#endif

	// State retrival
	cutilSafeCall( hipMemcpy(state_h, state_d, size_actual, hipMemcpyDeviceToHost) );

	return 0;
}
